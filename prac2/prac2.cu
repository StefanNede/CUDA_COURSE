#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2, a, b, c;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

__global__ void find_average(float *d_z, float *d_v) 
{
  float y1, value, sum, average;
  int   ind;

  // move array pointers to correct position

  ind = threadIdx.x + N*blockIdx.x*blockDim.x;
  sum = 0.0f;

  // average with random number calculation

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    // printf("Index: %d\n", ind);
    ind += blockDim.x;      // shift pointer to next element
    value = a*y1*y1 + b*y1 + c;
    sum += value;
  }

  average = sum/N;
  // printf("AVERAGE: %f\n", average);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = average;
}

__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // version 2
  // ind = 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    // printf("Index: %d\n", ind);
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    y2   = rho*y1 + alpha*d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array

  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
  // printf("VERSION 1\n");
    
  // NPATH = number of path simulations we are doing
  // N = number of timesteps 
  int     NPATH=9600000, h_N=200;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2, h_a, h_b, h_c;
  float  *h_v, *d_v, *d_z;
  double  sum1, sum2;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;
  h_a = 1.0f;
  h_b = 2.0f;
  h_c = 3.0f;

  // potentially could pass these in as a struct but not recommended on forums
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(a), &h_a, sizeof(h_a)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(b), &h_b, sizeof(h_b)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c), &h_c, sizeof(h_c)) );

  // random number generation

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );

  hipEventRecord(start); // for stopwatch
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, h_N*NPATH, 0.0f, 1.0f) );
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop); // get ellapsed time in milliseconds

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, h_N*NPATH/(0.001*milli));

  // execute kernel and time it -- Path simulations

  // hipEventRecord(start);
  // 128 threads per block so want NPATH/128 blocks as each thread is doing 1 and only 1 path
  // pathcalc<<<NPATH/128, 128>>>(d_z, d_v);
  // hipEventRecord(stop);

  // hipEventSynchronize(stop);
  // hipEventElapsedTime(&milli, start, stop);

  // getLastCudaError("pathcalc execution failed\n");
  // printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // execute kernel and time it -- Finding mean of normal random variables
  
  hipEventRecord(start);
  find_average<<<NPATH/128, 128>>>(d_z, d_v);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("find_average execution failed\n");
  printf("Monte Carlo kernel execution time (ms): %f \n",milli);


  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average + variance/std. dev

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    // printf("%f\n", h_v[i]);
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
